#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// CUDA Kernel: 保留 K 参数，但假设 K=1
__global__ void conv2d_kernel(float* input, float* weight, float* output, int H,
                              int W, int I, int J, int C, int K) {
  int Ho = H - I + 1;
  int Wo = W - J + 1;

  int h = blockIdx.y * blockDim.y + threadIdx.y;
  int w = blockIdx.x * blockDim.x + threadIdx.x;
  int k = blockIdx.z * blockDim.z + threadIdx.z;

  if (h < Ho && w < Wo && k < K) {  // K=1
    float sum = 0.0f;
    for (int i = 0; i < I; i++) {
      for (int j = 0; j < J; j++) {
        for (int c = 0; c < C; c++) {
          int in_idx = ((h + i) * W + (w + j)) * C + c;
          int wt_idx = ((i * J + j) * C + c) * K + k;
          sum += input[in_idx] * weight[wt_idx];
        }
      }
    }
    int out_idx = (h * Wo + w) * K + k;
    output[out_idx] = sum;
  }
}

void conv2d(float* input, float* weight, float* output, int H, int W, int I,
            int J, int C, int K) {
  int Ho = H - I + 1;
  int Wo = W - J + 1;

  size_t in_size = H * W * C * sizeof(float);
  size_t wt_size = I * J * C * K * sizeof(float);  // K=1 时仍合法
  size_t out_size = Ho * Wo * K * sizeof(float);   // K=1 时仍合法

  float *d_input, *d_weight, *d_output;
  hipMalloc(&d_input, in_size);
  hipMalloc(&d_weight, wt_size);
  hipMalloc(&d_output, out_size);

  hipMemcpy(d_input, input, in_size, hipMemcpyHostToDevice);
  hipMemcpy(d_weight, weight, wt_size, hipMemcpyHostToDevice);

  dim3 block(16, 16, 1);
  dim3 grid((Wo + block.x - 1) / block.x, (Ho + block.y - 1) / block.y,
            (K + block.z - 1) / block.z);  // K=1 → grid.z=1

  conv2d_kernel<<<grid, block>>>(d_input, d_weight, d_output, H, W, I, J, C, K);

  hipMemcpy(output, d_output, out_size, hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_weight);
  hipFree(d_output);
}

int main() {
  // 参数: H=7, W=56, I=3, J=3, C=32, K=1
  int H = 7, W = 56, I = 3, J = 3, C = 32, K = 32;
  int Ho = H - I + 1;
  int Wo = W - J + 1;

  float* input = (float*)malloc(H * W * C * sizeof(float));
  float* weight = (float*)malloc(I * J * C * K * sizeof(float));
  float* output = (float*)malloc(Ho * Wo * K * sizeof(float));

  for (int idx = 0; idx < H * W * C; idx++) input[idx] = 1.0f;
  for (int idx = 0; idx < I * J * C * K; idx++) weight[idx] = 0.1f;

  conv2d(input, weight, output, H, W, I, J, C, K);

  // for (int h = 0; h < Ho; h++) {
  //   for (int w = 0; w < Wo; w++) {
  //     int out_idx = (h * Wo + w) * K;
  //     printf("out[%d,%d,0] = %.2f\n", h, w, output[out_idx]);
  //   }
  // }

  free(input);
  free(weight);
  free(output);
  return 0;
}
