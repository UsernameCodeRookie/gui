// gemm_gpgpusim_cycle.cu
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK_CUDA(call) do { \
    hipError_t e = (call); \
    if (e!=hipSuccess) { \
        printf("%s:%d CUDA Error %s\n",__FILE__,__LINE__,hipGetErrorString(e)); \
        exit(1); \
    } \
} while(0)

#define TILE_M 16
#define TILE_N 16
#define TILE_K 16

// Pure kernel cycle version: C = A*B, MxK * KxN = MxN
__global__ void gemm_kernel(const float* __restrict__ A,
                            const float* __restrict__ B,
                            float* __restrict__ C,
                            int M, int N, int K) 
{
    __shared__ float sA[TILE_M][TILE_K];
    __shared__ float sB[TILE_K][TILE_N];

    int row = blockIdx.y * TILE_M + threadIdx.y;
    int col = blockIdx.x * TILE_N + threadIdx.x;

    float sum = 0.0f;

    // Loop over tiles
    for (int t=0; t<(K+TILE_K-1)/TILE_K; t++) {
        int tiled_k = t*TILE_K + threadIdx.x;

        // Load A tile
        if(row<M && tiled_k<K)
            sA[threadIdx.y][threadIdx.x] = A[row*K + tiled_k];
        else
            sA[threadIdx.y][threadIdx.x] = 0.f;

        // Load B tile
        int tiled_row = t*TILE_K + threadIdx.y;
        if(tiled_row<K && col<N)
            sB[threadIdx.y][threadIdx.x] = B[tiled_row*N + col];
        else
            sB[threadIdx.y][threadIdx.x] = 0.f;

        __syncthreads();

        // Compute tile
        #pragma unroll
        for(int k=0;k<TILE_K;k++)
            sum += sA[threadIdx.y][k]*sB[k][threadIdx.x];

        __syncthreads();
    }

    if(row<M && col<N)
        C[row*N + col] = sum;
}

// Host wrapper for GPGPU-Sim
void gemm_kernel_test(float* A, float* B, float* C, int M, int N, int K)
{
    float *dA, *dB, *dC;
    size_t sizeA = M*K*sizeof(float);
    size_t sizeB = K*N*sizeof(float);
    size_t sizeC = M*N*sizeof(float);

    CHECK_CUDA(hipMalloc(&dA,sizeA));
    CHECK_CUDA(hipMalloc(&dB,sizeB));
    CHECK_CUDA(hipMalloc(&dC,sizeC));

    CHECK_CUDA(hipMemcpy(dA,A,sizeA,hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dB,B,sizeB,hipMemcpyHostToDevice));

    dim3 block(TILE_N,TILE_M); // 16x16 threads = 256 threads per block
    dim3 grid((N+TILE_N-1)/TILE_N,(M+TILE_M-1)/TILE_M);

    // Launch kernel
    gemm_kernel<<<grid,block>>>(dA,dB,dC,M,N,K);

    // Wait for kernel finish
    CHECK_CUDA(hipDeviceSynchronize());

    // Optional: don't copy back to host if purely measuring kernel cycle
    //CHECK_CUDA(cudaMemcpy(C,dC,sizeC,cudaMemcpyDeviceToHost));

    // Free device memory
    CHECK_CUDA(hipFree(dA));
    CHECK_CUDA(hipFree(dB));
    CHECK_CUDA(hipFree(dC));
}

int main()
{
    int M=64,N=64,K=64;

    size_t sizeA=M*K;
    size_t sizeB=K*N;
    size_t sizeC=M*N;

    float* A = (float*)malloc(sizeA*sizeof(float));
    float* B = (float*)malloc(sizeB*sizeof(float));
    float* C = (float*)malloc(sizeC*sizeof(float));

    for(size_t i=0;i<sizeA;i++) A[i]=1.f;
    for(size_t i=0;i<sizeB;i++) B[i]=1.f;
    for(size_t i=0;i<sizeC;i++) C[i]=0.f;

    // Run kernel-only version
    gemm_kernel_test(A,B,C,M,N,K);

    // Pure cycle test: 不打印、不 memcpy 回 host

    free(A); free(B); free(C);
    return 0;
}
